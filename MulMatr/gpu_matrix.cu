#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <utility>
#include <stdexcept>

#include "gpu_matrix.h"

void GPUMatrix::allocateMemory()
{
    hipMalloc(&_data, dataSize());

    hipMalloc(&_cudaData, sizeof(CudaMatrixData));
    CudaMatrixData data({ _height, _width, _data });
    hipMemcpy(_cudaData, &data, sizeof(CudaMatrixData), hipMemcpyHostToDevice);
}

void GPUMatrix::freeMemory()
{
    hipFree(_data);
    hipFree(_cudaData);
}

GPUMatrix::GPUMatrix(size_t height, size_t width) : BaseMatrix(height, width)
{
    allocateMemory();
}

GPUMatrix::GPUMatrix(const CPUMatrix &m) : GPUMatrix(m.height(), m.width())
{
    hipMemcpy(_data, m.data(), dataSize(), hipMemcpyHostToDevice);
}

GPUMatrix::GPUMatrix(const GPUMatrix &other) : GPUMatrix(other.height(), other.width())
{
    hipMemcpy(_data, other.data(), dataSize(), hipMemcpyDeviceToDevice);
}

GPUMatrix::GPUMatrix(GPUMatrix &&other) : BaseMatrix(std::move(other))
{
    _cudaData = other._cudaData;
    other._cudaData = nullptr;
}

GPUMatrix &GPUMatrix::operator=(const GPUMatrix &other)
{
    freeMemory();

    _height = other._height;
    _width = other._width;
    
    allocateMemory();

    hipMemcpy(_data, other.data(), dataSize(), hipMemcpyDeviceToDevice);

    return *this;
}

GPUMatrix::~GPUMatrix()
{
    freeMemory();
}

CPUMatrix GPUMatrix::toCPU()
{
    CPUMatrix output(height(), width());
    hipMemcpy(output.data(), _data, dataSize(), hipMemcpyDeviceToHost);
    return output;
}

double GPUMatrix::at(size_t i, size_t j) const
{
    double x;
    hipMemcpy(&x, _data + index(i, j), sizeof(double), hipMemcpyDeviceToHost);
    return x;
}

__global__ void mulMatrKernel(const CudaMatrixData *a, const CudaMatrixData *b,
    CudaMatrixData *res)
{
    size_t i = blockIdx.y * blockDim.y + threadIdx.y;
    size_t j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= res->height || j >= res->width)
        return;

    size_t index = i * res->width + j;
    res->data[index] = 0;
    for (size_t k = 0; k < a->width; k++)
        res->data[index] += a->data[i * a->width + k] * b->data[k * b->width + j];
}

void GPUMatrix::callKernel(const dim3 &cudaBlocks, const dim3 &cudaThreads,
    const CudaMatrixData *a, const CudaMatrixData *b, CudaMatrixData *result) const
{
    mulMatrKernel<<<cudaBlocks, cudaThreads>>>(a, b, result);
}

GPUMatrix GPUMatrix::multiply(const GPUMatrix &other) const
{
    if (_width != other._height)
        throw std::runtime_error("Wrong matrtix dimensions: a.width != b.height");

    GPUMatrix result = GPUMatrix(_height, other._width);

    static const unsigned int BLOCK_SIZE = 32;
    dim3 cudaThreads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 cudaBlocks(
        static_cast<unsigned int>(result._width + cudaThreads.x - 1) / cudaThreads.x,
        static_cast<unsigned int>(result._height + cudaThreads.y - 1) / cudaThreads.y
    );

    hipStream_t stream = 0;
    hipEvent_t endEvent;
    hipEventCreate(&endEvent);

    callKernel(cudaBlocks, cudaThreads, _cudaData, other._cudaData, result._cudaData);

    hipEventRecord(endEvent, stream);
    hipEventSynchronize(endEvent);

    hipEventDestroy(endEvent);

    return result;
}

CPUMatrix GPUMatrix::multiply(const CPUMatrix &other) const
{
    return multiply(GPUMatrix(other)).toCPU();
}
