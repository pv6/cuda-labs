#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <utility>
#include <stdexcept>

#include "gpu_matrix.h"

GPUMatrix::GPUMatrix(const CPUMatrix &m) : GPUMatrix(m.height(), m.width())
{
    hipMemcpy(_data, m.data(), dataSize(), hipMemcpyHostToDevice);
}

GPUMatrix::GPUMatrix(const GPUMatrix &other) : GPUMatrix(other.height(), other.width())
{
    hipMemcpy(_data, other.data(), dataSize(), hipMemcpyDeviceToDevice);
}

GPUMatrix::GPUMatrix(GPUMatrix &&other) : BaseMatrix(std::move(other))
{
    _cudaData = other._cudaData;
    other._cudaData = nullptr;
}

GPUMatrix &GPUMatrix::operator=(const GPUMatrix &other)
{
    hipFree(_data);

    _height = other._height;
    _width = other._width;
    hipMalloc(reinterpret_cast<void **>(&_data), dataSize());
    hipMemcpy(_data, other.data(), dataSize(), hipMemcpyDeviceToDevice);

    return *this;
}

GPUMatrix::~GPUMatrix()
{
    hipFree(_data);
    hipFree(_cudaData);
}

CPUMatrix GPUMatrix::toCPU()
{
    CPUMatrix output(height(), width());
    hipMemcpy(output.data(), _data, dataSize(), hipMemcpyDeviceToHost);
    return output;
}

double GPUMatrix::at(size_t i, size_t j) const
{
    double x;
    hipMemcpy(&x, _data + index(i, j), sizeof(double), hipMemcpyDeviceToHost);
    return x;
}

__global__ void mulMatrKernel(const CudaMatrixData *a, const CudaMatrixData *b,
    CudaMatrixData *res)
{
    size_t i = blockIdx.y * blockDim.y + threadIdx.y;
    size_t j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= res->height || j >= res->width)
        return;

    size_t index = i * res->width + j;
    res->data[index] = 0;
    for (size_t k = 0; k < a->width; k++)
        res->data[index] += a->data[i * a->width + k] * b->data[k * b->width + j];
}

GPUMatrix GPUMatrix::multiply(const GPUMatrix &other) const
{
    if (_width != other._height)
        throw std::runtime_error("Wrong matrtix dimensions: a.width != b.height");

    GPUMatrix result = GPUMatrix(_height, other._width);

    static const unsigned int BLOCK_SIZE = 32;
    dim3 cudaThreads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 cudaBlocks(
        static_cast<unsigned int>(result._width + cudaThreads.x - 1) / cudaThreads.x,
        static_cast<unsigned int>(result._height + cudaThreads.y - 1) / cudaThreads.y
    );

    hipStream_t stream = 0;
    hipEvent_t endEvent;
    hipEventCreate(&endEvent);

    mulMatrKernel<<<cudaBlocks, cudaThreads>>>(_cudaData, other._cudaData, result._cudaData);

    hipEventRecord(endEvent, stream);
    hipEventSynchronize(endEvent);

    hipEventDestroy(endEvent);

    return result;
}

CPUMatrix GPUMatrix::multiply(const CPUMatrix &other) const
{
    return multiply(GPUMatrix(other)).toCPU();
}

GPUMatrix::GPUMatrix(size_t height, size_t width) : BaseMatrix(height, width)
{
    hipMalloc(&_data, dataSize());

    hipMalloc(&_cudaData, sizeof(CudaMatrixData));
    CudaMatrixData data({ _height, _width, _data });
    hipMemcpy(_cudaData, &data, sizeof(CudaMatrixData), hipMemcpyHostToDevice);
}
